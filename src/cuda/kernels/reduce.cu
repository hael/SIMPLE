#include "hip/hip_runtime.h"
/* based on NVIDIA reduction talk
 * https://developer.download.nvidia.com/compute/cuda/1.1-Beta/x86_website/projects/reduction/doc/reduction.pdf
 *
 */
/*
template <unsigned int blockSize>
__global__ void reduce_int(int *g_idata, int *g_odata, unsigned int n)
{
    extern __shared__ int sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockSize*2) + tid;
    unsigned int gridSize = blockSize*2*gridDim.x;
    sdata[tid] = 0;
    while (i < n) { sdata[tid] += g_idata[i] + g_idata[i+blockSize]; i += gridSize; }
    __syncthreads();
    if (blockSize >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
    if (blockSize >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
    if (blockSize >= 128) { if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }
    if (tid < 32) {
       if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
       if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
       if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
       if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
       if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
a       if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
    }
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}
*/
template <unsigned int blockSize>
__global__ void reduce_float(float *g_idata, float *g_odata, unsigned int n)
{
    extern __shared__ float sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (blockSize * 2) + tid;
    unsigned int gridSize = blockSize * 2 * gridDim.x;
    sdata[tid] = 0.;
    while(i < n) {
        sdata[tid] += g_idata[i] + g_idata[i + blockSize];
        i += gridSize;
    }
    __syncthreads();
    if(blockSize >= 512) {
        if(tid < 256) {
            sdata[tid] += sdata[tid + 256];
        } __syncthreads();
    }
    if(blockSize >= 256) {
        if(tid < 128) {
            sdata[tid] += sdata[tid + 128];
        } __syncthreads();
    }
    if(blockSize >= 128) {
        if(tid < 64) {
            sdata[tid] += sdata[tid + 64];
        } __syncthreads();
    }
    if(tid < 32) {
        if(blockSize >= 64) sdata[tid] += sdata[tid + 32];
        if(blockSize >= 32) sdata[tid] += sdata[tid + 16];
        if(blockSize >= 16) sdata[tid] += sdata[tid + 8];
        if(blockSize >= 8) sdata[tid] += sdata[tid + 4];
        if(blockSize >= 4) sdata[tid] += sdata[tid + 2];
        if(blockSize >= 2) sdata[tid] += sdata[tid + 1];
    }
    if(tid == 0) g_odata[blockIdx.x] = sdata[0];
}
