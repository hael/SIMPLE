#include "hip/hip_runtime.h"
/*
 *   -- SIMPLE addon
 *      Author: Frederic Bonnet, Date: 29th of April 2015
 *      Monash University
 *      April 2015
 *
 *      Routine which calculates the product (element wise) of 
 *      hipDoubleComplex matrix A and hipDoubleComplex matrix B. and takes the
 *      real part of the product and puts it into a matrix of type double
 *      C = hipCreal(x) * hipCreal(y) + hipCimag(x) * hipCimag(y)
 *      C = Re( A * conjg(B) ) element wise
 *
 *      Non Special case
 * @precisions normal z -> s d c
*/
#include "common_magma.h"
#include "commonblas_zz2d.h"
#include "simple_cuDoubleComplex.h"
#include "simple.h"

#if defined (CUDA) /*preprossing for the OPENCL environment */

/*
 * Kernel that takes element wise using the standar kernel
 * C = alpha*A*B + beta*C
*/
extern "C" __global__ void
main_zz2dgemm_kernel_N_N( double *C, 
			  const hipDoubleComplex *A, 
			  const hipDoubleComplex *B,
			  int m, int n,
			  int lda, int ldb, int ldc,
			  double alpha, double beta)
{

  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;

  C[i * ldc + j ] = cuDmul(alpha, cuReCCstarmul( A[ i * lda + j], B[ i * ldb + j] ) );

  __syncthreads();

}
/*
 * Kernel that takes element wise using the shared memory kernel
 * C = alpha*A*B + beta*C
*/
extern "C" __global__ void
shared_zz2dgemm_kernel_N_N( double *C, 
			  const hipDoubleComplex *A, 
			  const hipDoubleComplex *B,
			  int m, int n,
			  int lda, int ldb, int ldc,
			  double alpha, double beta)
{

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  __shared__ double C_tile[16][17];

  C_tile[threadIdx.x][threadIdx.y] = cuDmul(alpha, cuReCCstarmul( A[ i * lda + j], B[ i * ldb + j] ) );

  __syncthreads();

  C[i * ldc + j ] = C_tile[threadIdx.x][threadIdx.y];

}

extern "C" int
zz2dgemm_kernel_N_N( double *C, 
		     const hipDoubleComplex *A, 
		     const hipDoubleComplex *B, 
		     int m, int n, int k, 
		     int lda, int ldb, int ldc, 
		     double alpha, double beta)
{
  int rc = 0; //return code
  
  dim3 threads(16,16);
  dim3 grid(m/16+(m%16!=0),n/16+(n%16!=0));

  shared_zz2dgemm_kernel_N_N<<< grid, threads >>>(C, A, B, 
						m, n,
						lda, ldb, ldc,
						alpha, beta);

  //making sure that the GPU is synchronized with CPU before proceeding
  hipCtxSynchronize();
  return rc;
} /* End of zz2dgemm_kernel_N_N */

#endif /* CUDA */
