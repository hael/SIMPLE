
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "hipblas.h"

extern "C" int f_cublasCreate(hipblasHandle_t **handle)
{
    *handle = (hipblasHandle_t*)malloc(sizeof(hipblasHandle_t));
    return hipblasCreate(*handle);
}

extern "C" int f_cublasDgemm(hipblasHandle_t *handle,
                             hipblasOperation_t transa, hipblasOperation_t transb,
                             int m, int n, int k,
                             const double *alpha,
                             const double *A, int lda,
                             const double *B, int ldb,
                             const double *beta,
                             double *C, int ldc)
{
    return hipblasDgemm(*handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
}

extern "C" int f_cublasDgemmBatched(hipblasHandle_t *handle,
                                    hipblasOperation_t transa, hipblasOperation_t transb,
                                    int m, int n, int k,
                                    const double *alpha,
                                    const double **A, int lda,
                                    const double **B, int ldb,
                                    const double *beta,
                                    double **C, int ldc,
                                    int batch_count)
{
    return hipblasDgemmBatched(*handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc, batch_count);
}
extern "C" int f_cublasSgemm(hipblasHandle_t *handle,
                             hipblasOperation_t transa, hipblasOperation_t transb,
                             int m, int n, int k,
                             const float *alpha,
                             const float *A, int lda,
                             const float *B, int ldb,
                             const float *beta,
                             float *C, int ldc)
{
    return hipblasSgemm(*handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
}

extern "C" int f_cublasSgemmBatched(hipblasHandle_t *handle,
                                    hipblasOperation_t transa, hipblasOperation_t transb,
                                    int m, int n, int k,
                                    const float *alpha,
                                    const float **A, int lda,
                                    const float **B, int ldb,
                                    const float *beta,
                                    float **C, int ldc,
                                    int batch_count)
{
    return hipblasSgemmBatched(*handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc, batch_count);
}
extern "C" void f_cublasDestroy(hipblasHandle_t *handle)
{
    hipblasDestroy(*handle);
    free(handle);
}

extern "C" int f_cudaStreamCreate(hipStream_t **stream)
{
    *stream = (hipStream_t *) malloc(sizeof(hipStream_t));
    return hipStreamCreate(*stream);
}

extern "C" int f_cublasSetStream(hipblasHandle_t *handle, hipStream_t *streamid)
{
    return hipblasSetStream(*handle, *streamid);
}

extern "C" void f_cudaStreamDestroy(hipStream_t *stream)
{
    hipStreamDestroy(*stream);
}