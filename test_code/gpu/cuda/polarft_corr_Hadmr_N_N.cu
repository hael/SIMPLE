#include "hip/hip_runtime.h"
/*
 *   -- SIMPLE addon
 *      Author: Frederic Bonnet, Date: 25th of September 2015
 *      Monash University
 *      Spetember 2015
 *
 *      Routine which calculates the product (element wise) of 
 *      hipDoubleComplex matrix A and hipDoubleComplex matrix B. and takes the
 *      real part of the product and puts it into a matrix of type double
 *      C = hipCreal(x) * hipCreal(y) + hipCimag(x) * hipCimag(y)
 *      C = Re( A * conjg(B) ) element wise
 *
 *      Non Special case
 * @precisions normal z -> s d c
*/
#include "common_magma.h"
#include "commonblas_zz2d.h"
#include "simple_cuDoubleComplex.h"
#include "polarft_gpu.h"
#include "simple.h"

#define imin(a,b) (a<b?a:b)
//#define debug true
//#define debug_high false
//#define debug_write false
#if defined (CUDA) /*preprossing for the OPENCL environment */

/* testcode for the kernel code 
test_r<<<ceil(npart/16.0),16>>>(d_r, npart, nrot, nk, alpha);
*/
extern "C" __global__ void
test_r( float *d_r, int npart, int nrot, int nk,
        float alpha){
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  d_r[0] = i ; __syncthreads();}

/* doing the r product Re(A*conjg(B)) */
extern "C" __global__ void
rprod_3D_mat( float *C,
              const hipFloatComplex *A,
              const hipFloatComplex *B,
              int npart, int nrot, int nk,
              float alpha)
{

  int ipart = blockIdx.x * blockDim.x + threadIdx.x;
  int  irot = blockIdx.y * blockDim.y + threadIdx.y;
  int    ik = blockIdx.z * blockDim.z + threadIdx.z;

  if ( ipart < npart) {
    if (irot < nrot ){
      if (ik < nk ){
        C[(irot+nrot*ik)*npart+ipart ] =
          cuReCCstarmulf( A[(irot+nrot*ik)*npart+ipart], 
                          B[(irot+nrot*ik)*npart+ipart] );
      }
    }
  }
  __syncthreads();
}


/* doing the sum(a,b)sq product Re(A*conjg(A)) */
extern "C" __global__ void
absq_3D_mat( float *C,
             const hipFloatComplex *A,
             int npart, int nrot, int nk,
             float alpha)
{

  int ipart = blockIdx.x * blockDim.x + threadIdx.x;
  int  irot = blockIdx.y * blockDim.y + threadIdx.y;
  int    ik = blockIdx.z * blockDim.z + threadIdx.z;

  if ( ipart < npart) {
    if (irot < nrot ){
      if (ik < nk ){
        C[(irot+nrot*ik)*npart+ipart ] = cuReCCstarf( A[(irot+nrot*ik)*npart+ipart]);
      }
    }
  }
  __syncthreads();
}

/* summing the 3D matrix treating it a 1D vec */
extern "C" __global__ void
sum_1D( float *A, float *partial, int N) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int sharedIndex = threadIdx.x;
  //allocating the shared memory array
  __shared__ float shared_A[256];

  //summing over the 
  float temp = 0.0;
  while ( i < N ){
    temp += A[i];
    i += blockDim.x * gridDim.x;
  } 
  shared_A[sharedIndex] = temp;
  //syncronizing the threads
  __syncthreads();

  //practising the 1/2 reducion sum at each step
  int j = blockDim.x / 2.0;
  while ( j != 0 ) {
    if ( sharedIndex < j ) shared_A[sharedIndex] += shared_A[sharedIndex + j];
    __syncthreads();
    j /= 2;
  }
  if ( sharedIndex == 0 ) partial[blockIdx.x] = shared_A[0];
}

/* summing test 1D case */
extern "C" int
test_sum_1D(int npart, int nrot, int nk) {
  int rc = 0;
  /* the error handlers from the cuda library */
  hipError_t err;

  float init_constant = 1.1234; //initial constant for h_A[N]

  int N = npart *nrot * nk;
  int threadsPerBlock = 256;
  int blocksPerGrid = imin(32,(N+threadsPerBlock-1)/threadsPerBlock);

  printf("N=%i, threadsPerBlock=%i, blocksPerGrid=%i\n",N,threadsPerBlock,blocksPerGrid);

  float c;
  int size_p_c = blocksPerGrid*sizeof(float);
  float *partial_c = (float*)malloc(size_p_c);
  int size_h_A = N*sizeof(float);
  float *h_A = (float*)malloc(size_h_A);
  for ( int i = 0 ; i<N ; i++ ){h_A[i] = init_constant;} //printf("h_A[%i]=%f\n",i,h_A[i]);}

  float *d_A1D = NULL;
  err = hipMalloc((void**)&d_A1D, size_h_A);
  err = hipMemcpy(d_A1D, h_A, N*sizeof(float), hipMemcpyHostToDevice);

  float *d_partial_c = NULL;
  err = hipMalloc((void**)&d_partial_c, size_p_c);

  sum_1D<<<blocksPerGrid,threadsPerBlock>>>(d_A1D,d_partial_c,N);
  err = hipMemcpy(partial_c, d_partial_c, size_p_c, hipMemcpyDeviceToHost);
  if ( (int)err != hipSuccess ) {rc =  get_error_id_corr_Hadmr_gpu(err);}

  c = 0.0;
  for ( int igrid = 0 ; igrid < blocksPerGrid ; igrid++ ) {  
    c += partial_c[igrid];
  }

  printf(ANSI_COLOR_BRIGHT_YELLOW"init_constant = %f, sum of vector A[%i], c= %f, at Line %i %s\n" ANSI_COLOR_RESET, init_constant, N, c, __LINE__,__FUNCTION__);

  free(h_A);
  free(partial_c);
  hipFree(d_A1D);
  hipFree(d_partial_c);

  return rc;
}

/* main kernel entry */
extern "C" int
polarft_corr_N_N(deviceDetails_t * s_devD,
                 polar_corr_calc_t *s_polar,
                 float *r,
                 const hipFloatComplex *A,
                 const hipFloatComplex *B,
                 int npart, int nrot, int nk,
                 float alpha,
                 bench_t *s_bench, debug_gpu_t *s_debug_gpu)
{
  int rc = 0;
  /*numbers of points to be considered */
  int npts = npart * nrot * nk;
  /* grid and threads block definition */
  int nx, ny, nz;          //Dimension of the threads
  int gridx, gridy, gridz; //Dimension of the 3D Grid
  /* setting the values into the object */
  pfts_Sizes *p_pfts_Sizes = new pfts_Sizes(npart,nrot,nk);
  mesh_3D *p_mesh_3D = new mesh_3D(s_polar);
  //TODO: fix the template argument when done
  //mesh_1D *p_mesh_1D = new mesh_1D(s_polar);
  /* the error handlers from the cuda library */
  hipError_t err;
  /*device allocation */
  float *d_r;
  hipFloatComplex *d_A;
  hipFloatComplex *d_B;
  float *d_C;   //device pointer for the r product
  /* size of the element in consideration */
  int size_r;
  int size_m;
  int size_reC;
  /* indexer */
  int i,j,k;
  /*start of the execution commands */
  hipFloatComplex *C = (hipFloatComplex*)malloc(npts);

  float *reC = (float*)malloc(npts);

  float suma;
  double sumb;

  /* nx=16; ny=16; nz=4; dim3 threads(16,16,4); */
  nx=s_polar->nx; ny=s_polar->ny; nz=s_polar->nz;
  dim3 threads(nx,ny,nz);
  gridx = npart/(float)nx+(npart%nx!=0);
  gridy =  nrot/(float)ny+( nrot%ny!=0);
  gridz =    nk/(float)nz+(   nk%nz!=0);
  dim3 grid(gridx,gridy,gridz);
  /*dim3 grid(npart/16.0+(npart%16!=0),
             nrot/16.0+( nrot%16!=0),
               nk/4.0+(     nk%4!=0) );
  */
  if (s_debug_gpu->debug_i == true ) {

    rc = print_3D_mesh(0,p_mesh_3D,s_polar,p_pfts_Sizes,nx,ny,nz);

    /* printing input variables on screen */
    if ( s_debug_gpu->debug_high_i == true ) {
      if ( s_debug_gpu->debug_write_i == true ) {
        rc = print_function_header_N_N_info(r,
                                            C, A, B, 
                                            npart, nrot, nk,
                                            alpha);
      }
    }
  }
  //allocating the memory on GPU
  size_r = 1 * sizeof(float);
  err = hipMalloc((void**)&d_r, size_r); if ( (int)err != hipSuccess ) {rc = err;}
  size_m = npts*sizeof(hipFloatComplex);
  err = hipMalloc((void**)&d_A, size_m);
  err = hipMalloc((void**)&d_B, size_m);
  size_reC = npts*sizeof(float);
  err = hipMalloc((void**)&d_C, size_reC);
  //uploading the matrices A, B to GPU
  err = hipMemcpy(d_A, A, npts*sizeof(hipFloatComplex), hipMemcpyHostToDevice);
  err = hipMemcpy(d_B, B, npts*sizeof(hipFloatComplex), hipMemcpyHostToDevice);
  //computing the r product
  rprod_3D_mat<<<grid,threads>>>( d_C, d_A, d_B, npart, nrot, nk, alpha);
  //synchronizing CPU with GPU
  hipCtxSynchronize();

  //now summing the r 
  int N = npart *nrot * nk;
  int threadsPerBlock = 256; //threads/per block=16*16 for more //lism
  int blocksPerGrid = imin(32,(N+threadsPerBlock-1)/threadsPerBlock);
  if ( s_debug_gpu->debug_i == true ) {
    rc = print_1D_mesh(0, NULL, N, threadsPerBlock, blocksPerGrid); }
  float c;
  int size_p_c = blocksPerGrid*sizeof(float);
  // summed value
  float *partial_c = (float*)malloc(size_p_c);
  float *d_partial_c = NULL;
  err = hipMalloc((void**)&d_partial_c, size_p_c);

  sum_1D<<<blocksPerGrid,threadsPerBlock>>>(d_C,d_partial_c,N);
  hipCtxSynchronize();
  err = hipMemcpy(partial_c, d_partial_c, size_p_c, hipMemcpyDeviceToHost);

  c = 0.0;
  for ( int igrid = 0 ; igrid < blocksPerGrid ; igrid++ ) {  
    c += partial_c[igrid];
  }
  s_polar->r_polar = c;
  free(partial_c);
  hipFree(d_partial_c);
  err = hipFree(d_C);

  //getting the PFT1 * conjg(PFT1)
  float *d_CaT;  //device pointer for for sumasq
  float *partial_ca = (float*)malloc(size_p_c);
  float *d_partial_ca = NULL;
  err = hipMalloc((void**)&d_partial_ca, size_p_c);
  err = hipMalloc((void**)&d_CaT, size_reC);

  absq_3D_mat<<<grid,threads>>>( d_CaT, d_A, npart, nrot, nk, alpha);
  //synchronizing CPU with GPU
  hipCtxSynchronize();
  
  if ( s_debug_gpu->debug_i == true ) {
    if ( s_debug_gpu->debug_high_i == true ) {
      float *h_CaT = (float*)malloc(size_reC);
      err = hipMemcpy(h_CaT, d_CaT, size_reC, hipMemcpyDeviceToHost);
  
      FILE * timeFile;
      timeFile = fopen("AAstar_gpu_CUDA.log","w");
      suma = 0.0;
      sumb = 0.0;
      for (i=0; i<npart ; i++) {
        for (j=0; j<nrot ; j++) {
          for (k=0; k<nk ; k++) {
            if ( s_debug_gpu->debug_write_i == true ) {
              fprintf(timeFile,"%i %i %i %20.8f\n",i,j,k, h_CaT[(j+nrot*k)*npart+i]);
            }
            suma += h_CaT[(j+nrot*k)*npart+i];
            sumb += (double)h_CaT[(j+nrot*k)*npart+i];
          }
        }
      }
  
      printf(ANSI_COLOR_BRIGHT_CYAN"Sum of h_CaT : "
             ANSI_COLOR_BRIGHT_YELLOW"single precision suma=%f, "
             ANSI_COLOR_BRIGHT_GREEN "double precision sumb=%f\n" ANSI_COLOR_RESET,
             suma, sumb);

      fclose(timeFile);
    }
  }
  
  sum_1D<<<blocksPerGrid,threadsPerBlock>>>(d_CaT,d_partial_ca,N);
  hipCtxSynchronize(); //synchronizing CPU with GPU
  err = hipMemcpy(partial_ca, d_partial_ca, size_p_c, hipMemcpyDeviceToHost);
  c = 0.0;
  for ( int igrid = 0 ; igrid < blocksPerGrid ; igrid++ ) {  
    c += partial_ca[igrid];
  }
  s_polar->sumasq_polar = c;

  free(partial_ca);
  hipFree(d_partial_ca);
  err = hipFree(d_CaT);

  //getting the PFT2 * conjg(PFT2)
  float *d_CbT;  //device pointer for for sumasq
  float *partial_cb = (float*)malloc(size_p_c);
  float *d_partial_cb = NULL;
  err = hipMalloc((void**)&d_partial_cb, size_p_c);
  err = hipMalloc((void**)&d_CbT, size_reC);

  absq_3D_mat<<<grid,threads>>>( d_CbT, d_B, npart, nrot, nk, alpha);
  hipCtxSynchronize();
  //synchronizing CPU with GPU
  sum_1D<<<blocksPerGrid,threadsPerBlock>>>(d_CbT,d_partial_cb,N);
  hipCtxSynchronize();
  err = hipMemcpy(partial_cb, d_partial_cb, size_p_c, hipMemcpyDeviceToHost);
  c = 0.0;
  for ( int igrid = 0 ; igrid < blocksPerGrid ; igrid++ ) {  
    c += partial_cb[igrid];
  }
  s_polar->sumbsq_polar = c;

  free(partial_cb);
  hipFree(d_partial_cb);
  err = hipFree(d_CbT);

  /* summing test 1D case */
  if ( s_debug_gpu->debug_i == true ) {
    rc = print_s_polar_struct(s_polar);
    if ( s_debug_gpu->debug_high_i == true ) {
      test_sum_1D(npart, nrot, nk);
    }
  }

  /*freeing memory on device */
  err = hipFree(d_r);
  err = hipFree(d_A);
  err = hipFree(d_B);

  /* making sure that the GPU is synchronized with CPU before proceeding */
  hipCtxSynchronize();
  /* calling the destructor for the pfts_Sizes object */
  p_mesh_3D->~mesh_3D();
  //  p_mesh_1D->~mesh_1D();

  return rc;
 
} /* End of polarft_corr_N_N */

#endif /* CUDA */
