#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <errno.h>

#include <hip/hip_complex.h>
#include "hip/hip_runtime.h"


// Use hipCmulf from hip/hip_complex.h

/* Define CUDA kernel that squares the input complex array */
__global__ void  mul2DComplex(hipFloatComplex *in1,hipFloatComplex *in2, hipFloatComplex *out, int Nsize, int Msize)
{

  unsigned int n = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int m = blockIdx.y * blockDim.y + threadIdx.y;

  if(n > Nsize || m > Msize) return;
  int grid_width = gridDim.x * blockDim.x;
  int index = m + (n * grid_width);

  out[index] = hipCmulf (in1[index] , in2[index]);

}
// math_functions in hip/hip_runtime.h
__global__ void  mul2DFloat(float *in1,float *in2, float *out, int Nsize, int Msize)
{

  unsigned int n = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int m = blockIdx.y * blockDim.y + threadIdx.y;

  if(n > Nsize || m > Msize) return;
  int grid_width = gridDim.x * blockDim.x;
  int index = m + (n * grid_width);

  out[index] = (in1[index] * in2[index]);

}


/*
   Fortran subroutine arguments are passed by references.
   call fun( array_a, array_b, N) will be mapped to
   function (*a, *b, *N);
*/
extern "C"
{
#define KERNELMUL2DFLOAT kernelMul2DFloat_
void kernelMul2DFloat(float *a, float *b, float *c, int *Np, int *Ns, int *Bsize)
{
  int block_size=*Bsize;
  cuFloat *a_d,*b_d,*c_d;
  int N=*Np;int M=*Ns;
  hipSetDevice(0);

  /* Allocate complex array on device */
   hipMalloc ((void **) &a_d , sizeof(cuFloat)*N*M );
   hipMalloc ((void **) &b_d , sizeof(cuFloat)*N*M );
   hipMalloc ((void **) &c_d , sizeof(cuFloat)*N*M );
  // if(hipGetLastError() != hipSuccess){
  // printf("%s\n",hipGetErrorString(hipGetLastError()));
  // exit(1);
  // }

  /* Copy array from host memory to device memory */
  hipMemcpy( a_d, a,  sizeof(cuFloat)*N*M   ,hipMemcpyHostToDevice);
  hipMemcpy( b_d, b,  sizeof(cuFloat)*N*M   ,hipMemcpyHostToDevice);
  // if( hipMemcpy( a_d, a,  sizeof(hipFloatComplex)*N*M   ,hipMemcpyHostToDevice) != hipSuccess){
  // printf("%s\n",hipGetErrorString(hipGetLastError()));
  // exit(1);
  // }

  /* Compute execution configuration */
   dim3 dimBlock(block_size, 8);
   dim3 dimGrid ; //(N/dimBlock.x);
   dimGrid.x = (N + dimBlock.x - 1) / dimBlock.x;
   dimGrid.y = (M + dimBlock.y - 1) / dimBlock.y;


   if( N*N % block_size != 0 ) dimGrid.x+=1;

  /* Execute the kernel */
   mul2DFloat<<<dimGrid,dimBlock>>>(a_d,b_d,c_d,N,M);
  if( hipGetLastError()!= hipSuccess){
    printf("%s\n",hipGetErrorString(hipGetLastError()));
    exit(1);
  }

  /* Copy the result back */
  hipMemcpy( c, c_d, sizeof(hipFloatComplex)*N*M,hipMemcpyDeviceToHost);
  // if( hipGetLastError()!= hipSuccess){
  // printf("%s\n",hipGetErrorString(hipGetLastError()));
  // exit(1);
  // }

  /* Free memory on the device */
  hipFree(a_d, b_d, c_d);

  return;
}
#define KERNELMUL2DCOMPLEX kernelMul2DComplex_
void kernelmul2dcomplex_(hipFloatComplex *a, hipFloatComplex *b, hipFloatComplex *c, int *Np, int *Ns, int *Bsize)
{
  int block_size=*Bsize;
  hipFloatComplex *a_d,*b_d;
  int N=*Np;int M=*Ns;
  hipSetDevice(0);

  /* Allocate complex array on device */
   hipMalloc ((void **) &a_d , sizeof(hipFloatComplex)*N*M*2 );
   hipMalloc ((void **) &b_d , sizeof(hipFloatComplex)*N*M*2 );
  // if(hipGetLastError() != hipSuccess){
  // printf("%s\n",hipGetErrorString(hipGetLastError()));
  // exit(1);
  // }

  /* Copy array from host memory to device memory */
  hipMemcpy( a_d, a,  sizeof(hipFloatComplex)*N*M   ,hipMemcpyHostToDevice);
  hipMemcpy( b_d, b,  sizeof(hipFloatComplex)*N*M   ,hipMemcpyHostToDevice);
 // if( hipMemcpy( a_d, a,  sizeof(hipFloatComplex)*N*M   ,hipMemcpyHostToDevice) != hipSuccess){
  // printf("%s\n",hipGetErrorString(hipGetLastError()));
  // exit(1);
  // }

  /* Compute execution configuration */
   dim3 dimBlock(block_size, 8);
   dim3 dimGrid ; //(N/dimBlock.x);
   dimGrid.x = (N + dimBlock.x - 1) / dimBlock.x;
   dimGrid.y = (M + dimBlock.y - 1) / dimBlock.y;


   if( N*N % block_size != 0 ) dimGrid.x+=1;

  /* Execute the kernel */
  mul2DComplex<<<dimGrid,dimBlock>>>(a_d,b_d,b_d,N,M);
  // if( hipGetLastError()!= hipSuccess){
  // printf("%s\n",hipGetErrorString(hipGetLastError()));
  // exit(1);
  // }

  /* Copy the result back */
  hipMemcpy( c, b_d, sizeof(hipFloatComplex)*N*M,hipMemcpyDeviceToHost);
  // if( hipGetLastError()!= hipSuccess){
  // printf("%s\n",hipGetErrorString(hipGetLastError()));
  // exit(1);
  // }

  /* Free memory on the device */
  hipFree(a_d);

  return;
}

}
