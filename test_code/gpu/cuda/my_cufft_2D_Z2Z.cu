// includes, system

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
//include the cufft library
#include <hipfft/hipfft.h>

#define ANSI_COLOR_RED     "\x1b[31m"
#define ANSI_COLOR_GREEN   "\x1b[32m"
#define ANSI_COLOR_YELLOW  "\x1b[33m"
#define ANSI_COLOR_BLUE    "\x1b[34m"
#define ANSI_COLOR_MAGENTA "\x1b[35m"
#define ANSI_COLOR_CYAN    "\x1b[36m"
#define ANSI_COLOR_RESET   "\x1b[0m"

//declarations
void runtest(int argc, char**argv);

int main(int argc, char **argv) {
  runtest(argc,argv);
}
void runtest(int argc, char**argv) {

  int nx = 12000;
  int ny = 12000;

  int npts = nx * ny;
  /* the error handlers from the cuda library */
  hipError_t err;
  hipfftResult cufft_err;
  /* the plan for the cuFFT */
  hipfftHandle plan_fwd;
  hipfftHandle plan_bwd;

  printf("nx=%i, ny=%i, npts=%i, nx * ny=%i\n",nx, ny, npts, nx * ny);
  printf("nx*ny*sizeof(cufftDoubleComplex)=%lu\n",nx * ny*sizeof(hipfftDoubleComplex));

  hipfftDoubleComplex *h_in;
  hipfftDoubleComplex *h_out = (hipfftDoubleComplex*)malloc(sizeof(hipfftDoubleComplex) * npts);
  hipfftDoubleComplex *h_in_rev = (hipfftDoubleComplex*)malloc(sizeof(hipfftDoubleComplex) * npts);
  
  h_in = (hipfftDoubleComplex*)malloc(sizeof(hipfftDoubleComplex)* npts);
  for (unsigned int i=0 ; i < nx ; i++) {
    for (unsigned int j=0 ; j < ny ; j++) {
      h_in[i+nx*j].x = rand() / (float)RAND_MAX;
      h_in[i+nx*j].y = sin(i*4.0*atan(1.0)*2.0/npts);
    }
  }

  printf("pi: %f\n",4.0*atan(1.0));
  
  hipfftDoubleComplex *d_in;
  hipfftDoubleComplex *d_in_rev;
  hipfftDoubleComplex *d_out;

  hipMalloc((void**)&d_in, npts*sizeof(hipfftDoubleComplex));
  hipMalloc((void**)&d_out, npts*sizeof(hipfftDoubleComplex));
  hipMemcpy(d_in, h_in, npts * sizeof(hipfftDoubleComplex), hipMemcpyHostToDevice);

  //tranform data
  hipfftPlan2d(&plan_fwd, nx, ny, HIPFFT_Z2Z);
  hipfftExecZ2Z(plan_fwd, (hipfftDoubleComplex *)d_in, (hipfftDoubleComplex *)d_out, HIPFFT_FORWARD);
  //copy trans into h_out from device 
  hipMemcpy(h_out, d_out, npts*sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost);

  cufft_err = hipfftDestroy(plan_fwd);
  err = hipFree(d_in);

  //transform back
  hipMalloc((void**)&d_in_rev, npts*sizeof(hipfftDoubleComplex));
  hipfftPlan2d(&plan_bwd, nx, ny, HIPFFT_Z2Z);
  hipfftExecZ2Z(plan_bwd, (hipfftDoubleComplex *)d_out, (hipfftDoubleComplex *)d_in_rev, HIPFFT_BACKWARD);
  hipMemcpy(h_in_rev, d_in_rev, npts*sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost);

  cufft_err = hipfftDestroy(plan_bwd);
  err = hipFree(d_in_rev);
  err = hipFree(d_out);

  // check result
  for (unsigned int i = 0; i < nx-(nx-3); ++i) {
    for (unsigned int j = 0; j < ny-(ny-3); ++j)
      {
	h_out[i+nx*j].x = h_out[i+nx*j].x / (float)npts;
	h_out[i+nx*j].y /= (float)npts;

	h_in_rev[i+nx*j].x = h_in_rev[i+nx*j].x / (float)npts;
	h_in_rev[i+nx*j].y /= (float)npts;

	printf( ANSI_COLOR_GREEN "data: %15.8f %15.8f"
		ANSI_COLOR_BLUE" Fourier %15.8f %15.8f"
		ANSI_COLOR_RED" Inverse %15.8f %15.8f\n",
		h_in[i+nx*j].x, h_in[i+nx*j].y, 
		h_out[i+nx*j].x, h_out[i+nx*j].y, 
		h_in_rev[i+nx*j].x, h_in_rev[i+nx*j].y);

      }
  }

  free(h_in);
  free(h_out);
  free(h_in_rev);

  hipDeviceReset();
}
