#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <errno.h>

#include "hip/hip_complex.h"
#include "hip/hip_runtime.h"
#include "simple_cuda.h"

// Use hipCmulf from hip/hip_complex.h
/* Define CUDA kernel that squares the input complex array */
__global__ void  mul1DComplex(hipComplex *in1, hipComplex *in2, hipComplex *out, int N)
{
    unsigned int index   = blockIdx.x * blockDim.x + threadIdx.x;
    if(index < N) {
        out[index] = hipCmulf(in1[index], in2[index]);
    }

}
/* Define CUDA kernel that squares the input complex array */
__global__ void  mul2DComplex(hipComplex *in1, hipComplex *in2, hipComplex *out, int Nsize, int Msize)
{

    unsigned int n = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int m = blockIdx.y * blockDim.y + threadIdx.y;

    if(n > Nsize || m > Msize) return;
    int grid_width = gridDim.x * blockDim.x;
    unsigned long index = m + (n * grid_width);

    out[index] = hipCmulf(in1[index] , in2[index]);

}
/* Define CUDA kernel that squares the input complex array */
__global__ void  muladd2DComplex(hipComplex *in1, hipComplex *in2, hipComplex *out, int Nsize, int Msize)
{

    unsigned int n = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int m = blockIdx.y * blockDim.y + threadIdx.y;

    if(n > Nsize || m > Msize) return;
    int grid_width = gridDim.x * blockDim.x;
    unsigned long index = m + (n * grid_width);

    out[index] = hipCfmaf(in1[index] , in2[index],  out[index]);

}

// math_functions in hip/hip_runtime.h
__global__ void  mul2DFloat(float *in1, float *in2, float *out, int Nsize, int Msize)
{

    unsigned int n = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int m = blockIdx.y * blockDim.y + threadIdx.y;

    if(n > Nsize || m > Msize) return;
    int grid_width = gridDim.x * blockDim.x;
    unsigned long index = m + (n * grid_width);

    out[index] = (in1[index] * in2[index]);

}
// math_functions in hip/hip_runtime.h
__global__ void  muladd2DFloat(float *in1, float *in2, float *out, int Nsize, int Msize)
{

    unsigned int n = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int m = blockIdx.y * blockDim.y + threadIdx.y;

    if(n > Nsize || m > Msize) return;
    int grid_width = gridDim.x * blockDim.x;
    unsigned long index = m + (n * grid_width);

    out[index] = (in1[index] * in2[index]) + out[index];

}


/*
   Fortran subroutine arguments are passed by references.
   call fun( array_a, array_b, N) will be mapped to
   function (*a, *b, *N);
*/


// C=A*B
//#define KERNELMUL2DCOMPLEX kernelmul2dcomplex_
extern "C" void kernelmul2dcomplex_(hipComplex *a, hipComplex *b, hipComplex *c, int *Ncol, int *Nrow, int *Bsize)
{
    int block_size = *Bsize;
    hipComplex *a_d, *b_d, *c_d;
    int N = *Ncol; int M = *Nrow;
//  printf("In kernelmul2dcomplex matsize %d, %d, matsize (bytes) %lu, %lu, %lu\n", N,M, sizeof(a),sizeof(b),sizeof(c));

    hipSetDevice(0);

    /* Allocate complex array on device */
    hipMalloc((void **) &a_d , sizeof(hipComplex)*N * M);
    hipMalloc((void **) &b_d , sizeof(hipComplex)*N * M);
    hipMalloc((void **) &c_d , sizeof(hipComplex)*N * M);
    cudaCheckErrors("mul2d Malloc failed.");


    /* Copy array from host memory to device memory */
    hipMemcpy(a_d, a,  sizeof(hipComplex)*N * M   , hipMemcpyHostToDevice);
    hipMemcpy(b_d, b,  sizeof(hipComplex)*N * M   , hipMemcpyHostToDevice);
    cudaCheckErrors("mul2d Memcpy failed.");

    /* Compute execution configuration */
    dim3 dimBlock(block_size, 8);
    dim3 dimGrid ;//(N/dimBlock.x);
    dimGrid.x = (N + dimBlock.x - 1) / dimBlock.x;
    dimGrid.y = (M + dimBlock.y - 1) / dimBlock.y;

    if(N * M % block_size != 0) dimGrid.x += 1;
//printf("dimGrid %d,%d,%d\n",dimGrid.x,dimGrid.y,dimGrid.z);
//printf("dimBlock %d,%d,%d\n",dimBlock.x,dimBlock.y,dimBlock.z);
    /* Execute the kernel */
    mul2DComplex <<< dimGrid, dimBlock>>>(a_d, b_d, c_d, N, M);
    cudaCheckErrors("mul2d mul2DComplex failed.");
    /* Copy the result back */
    hipMemcpy(c, c_d, sizeof(hipComplex)*N * M, hipMemcpyDeviceToHost);
    cudaCheckErrors("mul2d Memcpy failed.");
    /* Free memory on the device */
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);

    return;
}

extern "C" void kernelmul1dcomplex_(hipComplex *a, hipComplex *b, hipComplex *c, int *Np)
{
    int block_size = 8;
    hipComplex *a_d, *b_d, *c_d;
    int N = *Np;
    hipSetDevice(0);

    /* Allocate complex array on device */
    hipMalloc((void **) &a_d , sizeof(hipComplex)*N);
    hipMalloc((void **) &b_d , sizeof(hipComplex)*N);
    hipMalloc((void **) &c_d , sizeof(hipComplex)*N);

    // if(hipGetLastError() != hipSuccess){
    // printf("%s\n",hipGetErrorString(hipGetLastError()));
    // exit(1);
    // }

    /* Copy array from host memory to device memory */
    hipMemcpy(a_d, a,  sizeof(hipComplex)*N   , hipMemcpyHostToDevice);
    hipMemcpy(b_d, b,  sizeof(hipComplex)*N   , hipMemcpyHostToDevice);
    cudaCheckErrors("mul1d Memcpy failed.");

    /* Compute execution configuration */
    dim3 dimBlock(block_size, 8);
    dim3 dimGrid(N / dimBlock.x);
    if(N % block_size != 0) dimGrid.x += 1;

    /* Execute the kernel */
    mul1DComplex <<< dimGrid, dimBlock>>>(a_d, b_d, c_d, N);
    cudaCheckErrors("mul1d mul1DComplex<> failed.");

    /* Copy the result back */
    hipMemcpy(c, c_d, sizeof(hipComplex)*N, hipMemcpyDeviceToHost);
    cudaCheckErrors("mul1d Memcpy failed.");

    /* Free memory on the device */
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);

    return;
}




// C=A*B
//#define KERNELMUL2DFLOAT kernelmul2dfloat_
extern "C" void kernelmul2dfloat_(float *a, float *b, float *c, int *Np, int *Ns, int *Bsize)
{
    int block_size = *Bsize;
    float *a_d, *b_d, *c_d;
    int N = *Np; int M = *Ns;
    hipSetDevice(0);

    /* Allocate complex array on device */
    hipMalloc((void **) &a_d , sizeof(float)*N * M);
    hipMalloc((void **) &b_d , sizeof(float)*N * M);
    hipMalloc((void **) &c_d , sizeof(float)*N * M);
    cudaCheckErrors("mul2df Malloc failed.");


    /* Copy array from host memory to device memory */
    hipMemcpy(a_d, a,  sizeof(float)*N * M   , hipMemcpyHostToDevice);
    hipMemcpy(b_d, b,  sizeof(float)*N * M   , hipMemcpyHostToDevice);
    cudaCheckErrors("mul2df Memcpy failed.");


    /* Compute execution configuration */
    dim3 dimBlock(block_size, 8);
    dim3 dimGrid ; //(N/dimBlock.x);
    dimGrid.x = (N + dimBlock.x - 1) / dimBlock.x;
    dimGrid.y = (M + dimBlock.y - 1) / dimBlock.y;


    if(N * N % block_size != 0) dimGrid.x += 1;

    /* Execute the kernel */
    mul2DFloat <<< dimGrid, dimBlock>>>(a_d, b_d, c_d, N, M);
    cudaCheckErrors("mul2df kernel failed.");

    /* Copy the result back */
    hipMemcpy(c, c_d, sizeof(float)*N * M, hipMemcpyDeviceToHost);
    cudaCheckErrors("mul2df Memcpy failed.");

    /* Free memory on the device */
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);

    return;
}

// D=A*B + C
//#define KERNELMULADD2DCOMPLEX kernelmul2dcomplex_
extern "C" void kernelmuladd2dcomplex_(hipComplex *a, hipComplex *b,hipComplex *c, hipComplex *d, int *Ncol, int *Nrow, int *Bsize)
{
    int block_size = *Bsize;
    hipComplex *a_d, *b_d, *c_d;
    int N = *Ncol; int M = *Nrow;
//  printf("In kernelmul2dcomplex matsize %d, %d, matsize (bytes) %lu, %lu, %lu\n", N,M, sizeof(a),sizeof(b),sizeof(c));

    hipSetDevice(0);

    /* Allocate complex array on device */
    hipMalloc((void **) &a_d , sizeof(hipComplex)*N * M);
    hipMalloc((void **) &b_d , sizeof(hipComplex)*N * M);
    hipMalloc((void **) &c_d , sizeof(hipComplex)*N * M);
    cudaCheckErrors("mul2d Malloc failed.");


    /* Copy array from host memory to device memory */
    hipMemcpy(a_d, a,  sizeof(hipComplex)*N * M   , hipMemcpyHostToDevice);
    hipMemcpy(b_d, b,  sizeof(hipComplex)*N * M   , hipMemcpyHostToDevice);
    hipMemcpy(c_d, c,  sizeof(hipComplex)*N * M   , hipMemcpyHostToDevice);

    cudaCheckErrors("mul2d Memcpy failed.");

    /* Compute execution configuration */
    dim3 dimBlock(block_size, 8);
    dim3 dimGrid ;//(N/dimBlock.x);
    dimGrid.x = (N + dimBlock.x - 1) / dimBlock.x;
    dimGrid.y = (M + dimBlock.y - 1) / dimBlock.y;

    if(N * M % block_size != 0) dimGrid.x += 1;
//printf("dimGrid %d,%d,%d\n",dimGrid.x,dimGrid.y,dimGrid.z);
//printf("dimBlock %d,%d,%d\n",dimBlock.x,dimBlock.y,dimBlock.z);
    /* Execute the kernel */
    muladd2DComplex <<< dimGrid, dimBlock>>>(a_d, b_d, c_d, N, M);
    cudaCheckErrors(" muladd2DComplex failed.");
    /* Copy the result back to D (not C)*/
    hipMemcpy(d, c_d, sizeof(hipComplex)*N * M, hipMemcpyDeviceToHost);
    cudaCheckErrors("mul2d Memcpy failed.");
    /* Free memory on the device */
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);

    return;
}
